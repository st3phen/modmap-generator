
#include <hip/hip_runtime.h>
//the Manhattan distance kernel - N is the side length of the images
__global__ void manhattan(const int *vect1, const int *vect2, int *out, int N){
	int xIndex = blockDim.x*blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y*blockIdx.y + threadIdx.y;
	
	if (xIndex >= N || yIndex >= N)
		return;
	
	int index = N*yIndex + xIndex;
	out[index] = abs(vect1[index] - vect2[index]);
}
