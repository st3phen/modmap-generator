
#include <hip/hip_runtime.h>
//the Euclidean distance kernel - N is the side length of the images
__global__ void euclidean(const int *vect1, const int *vect2, int *out, int N){
	int xIndex = blockDim.x*blockIdx.x + threadIdx.x;
    int yIndex = blockDim.y*blockIdx.y + threadIdx.y;
	
	if (xIndex >= N || yIndex >= N)
		return;
	
	int index = N*yIndex + xIndex;
	out[index] = (vect1[index] - vect2[index]) * (vect1[index] - vect2[index]);
}
